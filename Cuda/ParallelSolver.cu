#include "hip/hip_runtime.h"

#include <assert.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <Eigen/Dense>

std::tuple<Eigen::Map<Eigen::MatrixXd>,Eigen::Map<Eigen::VectorXi>> parallel_LU_pivot(Eigen::MatrixXd A);

int main(){
    Eigen::MatrixXd m(2,2);
    m(0,0) = 3;
    m(1,0) = 2.5;
    m(0,1) = -1;
    m(1,1) = m(1,0) + m(0,1);
    parallel_LU_pivot(m);

}
std::tuple<Eigen::Map<Eigen::MatrixXd>,Eigen::Map<Eigen::VectorXi>> parallel_LU_pivot(Eigen::MatrixXd A_){

    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;

    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;

    int lda = A_.rows(); //max dim of A (here rows)
    int n = A_.cols(); 
    int m = A_.rows();

    double *A = A_.data();
    // double *x = malloc(n*sizeof(double));
    int *Ipiv = (int*) malloc(lda*sizeof(int)); //Array containing pivot indizes
    double *LU =  (double*)malloc(n*m*sizeof(double));
    int info = 0;     /* host copy of error info */

    double *d_A = NULL; /* device copy of A */
    int *d_Ipiv = NULL; /* pivoting sequence */
    int *d_info = NULL; /* error info */
    int  lwork = 0;     /* size of workspace */
    double *d_work = NULL; /* device workspace for getrf */

    /* step 1: create cusolver handle, bind a stream */
    std::cout<<"1\n";
    status = hipsolverDnCreate(&cusolverH);
    std::cout<<status==HIPSOLVER_STATUS_NOT_INITIALIZED<<"2";
    // assert(HIPSOLVER_STATUS_SUCCESS == status);

    // cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    // assert(hipSuccess == cudaStat1);

    // std::cout<<"2\n";
    // status = hipsolverSetStream(cusolverH, stream);
    // assert(HIPSOLVER_STATUS_SUCCESS == status);

    // /* step 2: copy A to device */
    // cudaStat1 = hipMalloc ((void**)&d_A, sizeof(double) * n * m);
    // cudaStat3 = hipMalloc ((void**)&d_Ipiv, sizeof(int) * lda);
    // cudaStat4 = hipMalloc ((void**)&d_info, sizeof(int));
    // assert(hipSuccess == cudaStat1);
    // assert(hipSuccess == cudaStat2);
    // assert(hipSuccess == cudaStat3);
    // assert(hipSuccess == cudaStat4);

    // cudaStat1 = hipMemcpy(d_A, A, sizeof(double)*n*m, hipMemcpyHostToDevice);
    // assert(hipSuccess == cudaStat1);
    // assert(hipSuccess == cudaStat2);

 
    // /* step 3: query working space of getrf */
    // status = hipsolverDnDgetrf_bufferSize(
    //     cusolverH,
    //     m,
    //     n,
    //     d_A,
    //     lda,
    //     &lwork);
    // assert(HIPSOLVER_STATUS_SUCCESS == status);

    // cudaStat1 = hipMalloc((void**)&d_work, sizeof(double)*lwork);
    // assert(hipSuccess == cudaStat1);


    
    // status = hipsolverDnDgetrf(cusolverH,m,n, d_A, lda,d_work,d_Ipiv,d_info);
    // std::cout<<"3\n";
    // cudaStat1 = hipDeviceSynchronize();
    // assert(HIPSOLVER_STATUS_SUCCESS == status);
    // assert(hipSuccess == cudaStat1);
    // std::cout<<"4\n";
    // cudaStat1 = hipMemcpy(Ipiv , d_Ipiv, sizeof(int)*m, hipMemcpyDeviceToHost); 
    // cudaStat2 = hipMemcpy(LU   , d_A   , sizeof(double)*lda*m, hipMemcpyDeviceToHost);
    // cudaStat3 = hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
    // assert(hipSuccess == cudaStat1);
    // assert(hipSuccess == cudaStat2);
    // assert(hipSuccess == cudaStat3);


    // /*free GPU ressources*/
    // if (d_A) hipFree(d_A);
    // if (d_Ipiv) hipFree(d_Ipiv);
    // if (d_info) hipFree(d_info);
    // if (d_work) hipFree(d_work);
    // hipDeviceReset();

    // /*Transform array into Eigen::Matrix*/
    // Eigen::Map<Eigen::MatrixXd> LU_return(LU,m,n);
    // Eigen::Map<Eigen::VectorXi> Ipiv_return(Ipiv,m);

    // /*Free host data*/
    // if (A) free(A);

    // return std::make_tuple(LU_return,Ipiv_return);



}