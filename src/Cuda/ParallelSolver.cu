#include "hip/hip_runtime.h"
#include "ParallelSolver.hpp"


//Define device variables for stopping condition
__device__ unsigned int d_not_tolerent;
__device__  double d_marker;
__device__ unsigned int d_same;
__device__ unsigned int d_pos_of_same;

__global__ void reset_d_not_tolerent (){
    d_not_tolerent = 0;
}



//Calculate jacobi step for each element seperatly
__global__ void calc_jacobi_step(int n,double *A,double *b,double *x, double *residual){
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    double new_component=0;
    double zw = 0;
    for(int j=0;j<n;j++){
        zw += A[j*n+i]*x[j];
    }
    if(A[i*n+i]!= 0.0){
        new_component = (b[i]- zw)/A[i*n+i]+x[i];
        residual[i]=new_component-x[i];
    }else{
        residual[i]=0;
    }
    
}


//Check if solution has converged and update new solution
__global__ void update_and_check_tol(double *x, double *residual,double tol){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(std::abs(residual[i])>tol){
        if(d_marker !=residual[i] || d_pos_of_same != i){
            d_not_tolerent=1;
            d_marker = residual[i];
            d_pos_of_same =i;
            d_same =0;
        }else{
            if(d_same<10){
                d_not_tolerent=1;
            }
            d_same++;
        }
    }
    x[i]+=residual[i];
}


//external functions
namespace CUDA {

Eigen::VectorXd parallel_LU_pivot(Eigen::MatrixXd &A,Eigen::VectorXd &b){
    
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;


    const int m = A.cols();
    const int lda = A.cols();
    const int ldb = b.rows();
 
    Eigen::VectorXd x=Eigen::VectorXd::Zero(m); // x = A\B 
    int info = 0;     // host copy of error info 

    double *d_A = nullptr; // device copy of A 
    double *d_b = nullptr; // device copy of B 
    int *d_Ipiv = nullptr; // pivoting sequence
    int *d_info = nullptr; // error info 
    int  lwork = 0;     // size of workspace
    double *d_work = nullptr; // device workspace for getrf

    status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    assert(hipSuccess == cudaStat1);
    status = hipsolverSetStream(cusolverH, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    /////////////////
    // Copy to GPU //
    /////////////////
    cudaStat1 = hipMalloc ((void**)&d_A, sizeof(double) * lda * m);
    cudaStat2 = hipMalloc ((void**)&d_b, sizeof(double) * m);
    cudaStat3 = hipMalloc ((void**)&d_Ipiv, sizeof(int) * m);
    cudaStat4 = hipMalloc ((void**)&d_info, sizeof(int));
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);
    cudaStat1 = hipMemcpy(d_A, A.data(), sizeof(double)*lda*m, hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(d_b, b.data(), sizeof(double)*m, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);


    /////////////////////
    // Query workspace //
    /////////////////////
    status = hipsolverDnDgetrf_bufferSize(
        cusolverH,
        m,
        m,
        d_A,
        lda,
        &lwork);
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    cudaStat1 = hipMalloc((void**)&d_work, sizeof(double)*lwork);
    assert(hipSuccess == cudaStat1);

    //////////////////////
    // LU factorization //
    //////////////////////
    status = hipsolverDnDgetrf(
            cusolverH,
            m,
            m,
            d_A,
            lda,
            d_work,
            d_Ipiv,
            d_info);

    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cudaStat1);
    cudaStat1 = hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);



    /////////////////////
    //  solve A*x = b  //
    /////////////////////
    status = hipsolverDnDgetrs(
            cusolverH,
            HIPBLAS_OP_N,
            m,
            1, /* nrhs */
            d_A,
            lda,
            d_Ipiv,
            d_b,
            ldb,
            d_info);
    
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cudaStat1);
    cudaStat1 = hipMemcpy(x.data(), d_b, sizeof(double)*m, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    /////////////////////
    //  free recourses //
    /////////////////////
    if (d_A    ) hipFree(d_A);
    if (d_b    ) hipFree(d_b);
    if (d_Ipiv ) hipFree(d_Ipiv);
    if (d_info ) hipFree(d_info);
    if (d_work ) hipFree(d_work);

    if (cusolverH   ) hipsolverDnDestroy(cusolverH);
    if (stream      ) hipStreamDestroy(stream);

    hipDeviceReset();

    return x;
}




Eigen::VectorXd parallel_Jacobi_method(Eigen::MatrixXd &A,Eigen::VectorXd &b,double error){
    
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;
    hipError_t cudaStat5 = hipSuccess;
    hipError_t cudaStat6 = hipSuccess;
    hipError_t cudaStat7 = hipSuccess;
    int n = A.cols();
    Eigen::VectorXd x_0 = b;

    double *d_A = nullptr; // device copy of A 
    double *d_b = nullptr; // device copy of b
    double  *d_x = nullptr; // iterative solution
    double  *d_residual = nullptr;
    bool  *d_isfinished = nullptr;
    bool  *d_component_finished =nullptr;


    /////////////////
    // Copy to GPU //
    /////////////////
    cudaStat1 = hipMalloc (&d_A, sizeof(double)*n*n);
    cudaStat2 = hipMalloc (&d_b, sizeof(double)*n);
    cudaStat3 = hipMalloc (&d_x, sizeof(double)*n);
    cudaStat4 = hipMalloc (&d_residual, sizeof(double)*n);
    cudaStat5 = hipMalloc (&d_isfinished, sizeof(bool));
    // cudaStat6 = hipMalloc (&d_n, sizeof(int));
    cudaStat7 = hipMalloc (&d_component_finished, sizeof(bool)*n);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);
    assert(hipSuccess == cudaStat5);
    assert(hipSuccess == cudaStat6);
    assert(hipSuccess == cudaStat7);
    cudaStat1 = hipMemcpy(d_A, A.data(), sizeof(double)*n*n, hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(d_b, b.data(), sizeof(double)*n, hipMemcpyHostToDevice);
    cudaStat3 = hipMemcpy(d_x, x_0.data(), sizeof(double)*n, hipMemcpyHostToDevice);
    // cudaStat4 = hipMemcpy(d_x, &n, sizeof(int), hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);

    int blockSize = 16; //best performance for 16 threads
    int numBlocks = (n + blockSize - 1) / blockSize;
    int stop_after =100000;
    int counter = 0;
    typeof(d_not_tolerent) h_not_tolerent=1;
    ////////////////////////////
    // Calculate jacobi steps //
    ////////////////////////////
    while(counter < stop_after && h_not_tolerent){
        calc_jacobi_step<<<numBlocks, blockSize>>>(n,d_A,d_b,d_x,d_residual);
        update_and_check_tol<<<numBlocks, blockSize>>>(d_x, d_residual, error);

        if(counter%10 ==0){
            hipMemcpyFromSymbol(&h_not_tolerent, HIP_SYMBOL(d_not_tolerent), sizeof(d_not_tolerent)); 
            reset_d_not_tolerent<<<1, 1>>>(); 
        }
        counter++;
    }

    //Copy solution to host
    cudaStat1 = hipMemcpy(x_0.data(), d_x, sizeof(double)*n, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    /////////////////////
    //  free recourses //
    /////////////////////
    if (d_A    ) hipFree(d_A);
    if (d_b    ) hipFree(d_b);
    if (d_x    ) hipFree(d_x);
    if (d_residual    ) hipFree(d_residual);
    if (d_isfinished    ) hipFree(d_isfinished);
    if (d_component_finished    ) hipFree(d_component_finished);

    hipDeviceReset();

    return x_0;
}



}


