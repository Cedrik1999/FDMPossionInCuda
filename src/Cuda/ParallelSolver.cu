

#include "ParallelSolver.hpp"

namespace CUDA {

Eigen::VectorXd parallel_LU_pivot(Eigen::MatrixXd A,Eigen::VectorXd b){
    
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;


    const int m = A.cols();
    const int lda = A.cols();
    const int ldb = b.rows();
 
    Eigen::VectorXd x=Eigen::VectorXd::Zero(m); // x = A\B 
    int info = 0;     // host copy of error info 

    double *d_A = NULL; // device copy of A 
    double *d_b = NULL; // device copy of B 
    int *d_Ipiv = NULL; // pivoting sequence
    int *d_info = NULL; // error info 
    int  lwork = 0;     // size of workspace
    double *d_work = NULL; // device workspace for getrf

    status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    assert(hipSuccess == cudaStat1);
    status = hipsolverSetStream(cusolverH, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    /////////////////
    // Copy to GPU //
    /////////////////
    cudaStat1 = hipMalloc ((void**)&d_A, sizeof(double) * lda * m);
    cudaStat2 = hipMalloc ((void**)&d_b, sizeof(double) * m);
    cudaStat3 = hipMalloc ((void**)&d_Ipiv, sizeof(int) * m);
    cudaStat4 = hipMalloc ((void**)&d_info, sizeof(int));
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);
    cudaStat1 = hipMemcpy(d_A, A.data(), sizeof(double)*lda*m, hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(d_b, b.data(), sizeof(double)*m, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);


    /////////////////////
    // Query workspace //
    /////////////////////
    status = hipsolverDnDgetrf_bufferSize(
        cusolverH,
        m,
        m,
        d_A,
        lda,
        &lwork);
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    cudaStat1 = hipMalloc((void**)&d_work, sizeof(double)*lwork);
    assert(hipSuccess == cudaStat1);

    //////////////////////
    // LU factorization //
    //////////////////////
    status = hipsolverDnDgetrf(
            cusolverH,
            m,
            m,
            d_A,
            lda,
            d_work,
            d_Ipiv,
            d_info);

    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cudaStat1);
    cudaStat1 = hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);



    /////////////////////
    //  solve A*x = b  //
    /////////////////////
    status = hipsolverDnDgetrs(
            cusolverH,
            HIPBLAS_OP_N,
            m,
            1, /* nrhs */
            d_A,
            lda,
            d_Ipiv,
            d_b,
            ldb,
            d_info);
    
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cudaStat1);
    cudaStat1 = hipMemcpy(x.data(), d_b, sizeof(double)*m, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    /////////////////////
    //  free recourses //
    /////////////////////
    if (d_A    ) hipFree(d_A);
    if (d_b    ) hipFree(d_b);
    if (d_Ipiv ) hipFree(d_Ipiv);
    if (d_info ) hipFree(d_info);
    if (d_work ) hipFree(d_work);

    if (cusolverH   ) hipsolverDnDestroy(cusolverH);
    if (stream      ) hipStreamDestroy(stream);

    hipDeviceReset();

    return x;
}

}